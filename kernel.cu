#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cstdio>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <stdlib.h>
#include <assert.h>
#include <stdio.h>
#include <math.h>
#include <string>
#include <conio.h>
#include <stddef.h>
#include <fstream>
#include <sstream>
#include <tchar.h>

#include <gmp_util.h>

typedef long big_int;
using mpfr::mpreal;

__shared__ int dimension, ciclo, x, cram;
mpfr::mpreal temp, aux, temp1, det00;

std::vector < std::vector <mpreal> > matrixChi;
std::vector < std::vector <mpreal> > matrixChi2;

void input(mpreal array[][3], mpreal array1[][1]);
mpreal determinent(mpreal array[][3]);
mpreal calculate(mpreal array[][3], int a, int b, int c);
mpreal copy(mpreal array[][3], mpreal array1[][1], int a);
void comp_copy(mpreal array[][3], mpreal array1[][3]);
mpreal determinent4(mpreal array[][4]);
std::vector<std::vector<mpreal> > newdet(mpreal array[][4], int col);
void input4(mpreal array[][4], mpreal array1[][1]);
mpreal copy4(mpreal array[][4], mpreal array1[][1], int a);
void comp_copy4(mpreal array[][4], mpreal array1[][4]);

__host__ __device__ void f() {
#ifdef __CUDA_ARCH__
	printf("Hebra en CUDA: %d\n", threadIdx.x);
#else
	printf("CUDA Funcionando!\n");
#endif
}

__global__ void kernel() {
	f();
}


int main()
{

	const int digits = 200;
	mpreal::set_default_prec(mpfr::digits2bits(digits));
	mpreal overflow = std::numeric_limits<mpreal>::max();



	ciclo = 1;

	kernel << <1, 1 >> >();
	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "CUDA Fallo\n");
	}
	f();

	std::cout << "Corriendo variables con " << digits << " bits de largo\n\n";

	std::string s;
	x = 0;
	std::ifstream myReadFile;
	myReadFile.open("matrix.txt");
	if (myReadFile.is_open()) {
		myReadFile >> s;
		std::stringstream geek(s);
		dimension = 0;
		geek >> dimension;
		std::cout << "Dimensiones del problema: " << dimension << "\n";
	}
	myReadFile >> s;
	std::stringstream geek(s);
	cram = 0;
	geek >> cram;
	//generando CHI
	matrixChi.resize(dimension);
	for (int i = 0; i < dimension; i++)
	{
		matrixChi[i].resize(dimension + 1);
	}
	matrixChi2.resize(dimension - 1);
	for (int i = 0; i < dimension - 1; i++)
	{
		matrixChi2[i].resize(dimension);
	}
	//numeros desde file=matrix.txt

	if (myReadFile.is_open()) {
		for (int i = 0; i < dimension; i++) {
			for (int j = 0; j < dimension + 1; j++) {
				myReadFile >> s;
				std::stringstream geek(s);
				x = 0;
				geek >> x;
				matrixChi[i][j] = x;
			}
		}
	}
	myReadFile.close();
	for (int i = 0; i < dimension; i++) {
		for (int j = 0; j < dimension + 1; j++)
			if (j == dimension) std::cout << " | " << (mpreal)(matrixChi[i][j]) << " ";
			else std::cout << matrixChi[i][j] << " ";
			std::cout << "\n";
	}


	while (dimension > cram) {
		std::cout << "Generando Matrix de nivel " << dimension << "\n";


		//sacando pivote
		std::cout << "Pivote es " << matrixChi[0][0] << "\n";
		for (int i = 0; i < dimension - 1; i++)
			for (int j = 0; j < dimension; j++)
				matrixChi2[i][j] = (mpreal)(matrixChi[0][0] * matrixChi[i + 1][j + 1] - matrixChi[i + 1][0] * matrixChi[0][j + 1]);

		std::cout << "Condensacion a ciclo " << ciclo << "\n";
		ciclo++;
		for (int i = 0; i < dimension - 1; i++) {
			for (int j = 0; j < dimension; j++)
				if (j == dimension - 1) std::cout << " | " << (mpreal)(matrixChi2[i][j]) << " ";
				else std::cout << (mpreal)(matrixChi2[i][j]) << " ";
				std::cout << "\n";
		}

		//generando CHI
		dimension--;
		matrixChi.resize(dimension);
		for (int i = 0; i < dimension; i++)
		{
			matrixChi[i].resize(dimension + 1);
		}

		//poner datos de chi2 en chi
		for (int i = 0; i < dimension; i++)
			for (int j = 0; j < dimension + 1; j++)
				matrixChi[i][j] = (mpreal)matrixChi2[i][j];

		//generando la que se determina
		matrixChi2.resize(dimension - 1);
		for (int i = 0; i < dimension - 1; i++)
		{
			matrixChi2[i].resize(dimension);
		}
	}
	//comienza cramer

	std::cout << "\nDeterminantes de Cramer\n\n";

	bool sahi = true;
	while (sahi)
	{
		if (cram == 3 && cram != 4) {
			//long double permite valores de 1.8 � 10^308
			mpreal matrix[3][3];
			mpreal matrix1[3][1];
			mpreal reserve[3][3];
			mpreal detr[3];
			int sp1 = 0, teen = 1;
			int cont = 0;
			char in;
			//pasa los valores de matrixchi a la matriz de valores y resultados
			input(matrix, matrix1);
			//se respalda la matriz
			comp_copy(reserve, matrix);
			//se calcula el determinante general o coeficiente de la matriz
			det00 = determinent(matrix);
			while (sp1<3)
			{
				detr[cont] = copy(matrix, matrix1, sp1);
				comp_copy(matrix, reserve);
				cont++;
				sp1++;
			}
			cont = 0;
			while (cont<3)
			{
				std::cout << "x" << teen << " = " << (mpreal)detr[cont] << " /" << (mpreal)det00 << " [" << (mpreal)detr[cont] / (mpreal)det00 << "]" << std::endl;
				cont++;
				teen++;
			}
			mpreal x1 = ((mpreal)detr[0] / (mpreal)det00);
			mpreal x2 = ((mpreal)detr[1] / (mpreal)det00);
			mpreal x3 = ((mpreal)detr[2] / (mpreal)det00);
			mpreal err_total = 0;
			cont = 0;
			while (cont<3)
			{
				//calculo de errores
				mpreal resultado = matrix[cont][0] * x1 + matrix[cont][1] * x2 + matrix[cont][2] * x3;
				mpreal error = 100 - ((matrix1[cont][0] / resultado) * 100);
				std::cout << "[ESPERADO / OBTENIDO] para Ecuacion " << cont + 1 << "\n";
				std::cout << "[" << (mpreal)matrix1[cont][0] << " / " << (mpreal)resultado << "] - ";
				std::cout << "Error Relativo del " << abs((mpreal)error) << "%\n\n";
				err_total += abs((mpreal)error);
				cont++;
			}
			std::cout << "Error Promedio del " << (mpreal)err_total / 3 << "%\n";

			std::cout << "Finalizado, presione X para terminar\n" << overflow;
			std::cin >> in;
			if (in == 'x' || in == 'X')
				return 1;
		}
		else if (cram == 4) {
			mpreal matrix[4][4];
			mpreal matrix1[4][1];
			mpreal reserve[4][4];
			mpreal detr[4];
			int sp1 = 0, teen = 1;
			int cont = 0;
			char in;
			//pasa los valores de matrixchi a la matriz de valores y resultados
			input4(matrix, matrix1);
			//se respalda la matriz

			comp_copy4(reserve, matrix);
			//se calcula el determinante general o coeficiente de la matriz

			det00 = determinent4(matrix);

			while (sp1<4)
			{
				detr[cont] = copy4(matrix, matrix1, sp1);
				comp_copy4(matrix, reserve);
				cont++;
				sp1++;
			}
			cont = 0;
			while (cont<4)
			{
				std::cout << "x" << teen << " = " << (mpreal)detr[cont] << " /" << (mpreal)det00 << " [" << (mpreal)detr[cont] / (mpreal)det00 << "]" << std::endl;
				cont++;
				teen++;
			}
			mpreal x1 = ((mpreal)detr[0] / (mpreal)det00);
			mpreal x2 = ((mpreal)detr[1] / (mpreal)det00);
			mpreal x3 = ((mpreal)detr[2] / (mpreal)det00);
			mpreal x4 = ((mpreal)detr[3] / (mpreal)det00);
			mpreal err_total = 0;
			cont = 0;
			/*for (int k = 0; k<4; k++) {
				for (int l = 0; l<4; l++) {
					std::cout << matrix[k][l] << " ";
				}
				std::cout << "\n";
			}
			for (int l = 0; l<4; l++) {
				std::cout << matrix1[0][l] << "\n";
			}*/
			while (cont<4)
			{
				//calculo de errores
				/*std::cout << "test x1 " << matrix[cont][0] * x1 << "\n";
				std::cout << "test x2 " << matrix[cont][1] * x2 << "\n";
				std::cout << "test x3 " << matrix[cont][2] * x3 << "\n";
				std::cout << "test x4 " << matrix[cont][3] * x4 << "\n";
				std::cout << "test sum" << matrix[cont][0] * x1 + matrix[cont][1] * x2 + matrix[cont][2] * x3 + matrix[cont][3] * x4 << "\n";*/
				mpreal resultado = matrix[cont][0] * x1 + matrix[cont][1] * x2 + matrix[cont][2] * x3 + matrix[cont][3] * x4;
				/*std::cout << "Resultado " << resultado << "\n";*/
				mpreal error = 100 - ((matrix1[cont][0] / resultado) * 100);
				std::cout << "\n[ESPERADO / OBTENIDO] para Ecuacion " << cont + 1 << "\n";
				std::cout << "[" << (mpreal)matrix1[cont][0] << " / " << (mpreal)resultado << "] - ";
				std::cout << "Error Relativo del " << abs((mpreal)error) << "%\n\n";
				err_total += abs((mpreal)error);
				cont++;
			}
			std::cout << "Error Promedio del " << (float)err_total / 4 << "%\n";

			std::cout << "Finalizado, presione X para terminar\n";
			std::cin >> in;
			if (in == 'x' || in == 'X')
				return 1;
		}
	}
	std::cout.flush();
	return 0;
}
void input(mpreal array[][3], mpreal array1[][1])
{
	//traspasa los valores de la matriz X Y Z
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			array[i][j] = (mpreal)matrixChi[i][j];
		}
	}
	//traspasa los resultados de cada ecuacion X + Y + Z = RESULTADO
	for (int i = 0; i < 3; i++) {
		array1[i][0] = (mpreal)matrixChi[i][3];
	}
}
mpreal determinent(mpreal array[][3])
{
	int rows = 1, col = 1;
	int z = 0;
	temp = 0;
	int cont = 1;
	int x = 0;
	while (x<3)
	{
		//por cada columna los determinantes de 2x2 y los multiplica por l columna nueva dando el 3x3
		temp = temp + cont*(array[0][x] * calculate(array, rows, col, z));
		col = col * 0;
		z = z + cont;
		cont = cont*-1;
		x++;
	}
	if(cram == 3)
	std::cout << "\nDeterminante de la matrix 3x3 es " << temp << "\n\n";
	return temp;
}
mpreal calculate(mpreal array[][3], int a, int b, int c)
{
	//calcula los determinantes de 2x2
	temp1 = (array[a][b] * array[a + 1][b + 1 + c]) - (array[a + 1][b] * array[a][b + 1 + c]);
	return temp1;
}
mpreal copy(mpreal array[][3], mpreal array1[][1], int a)
{
	//traspasa los valores de la matriz de resultados a la matriz de valores
	int col = 0;
	temp = 0;
	while (col<3)
	{
		array[col][a] = array1[col][0];
		col++;
	}
	int i = 0, j = 0;
	while (i<3)
	{
		j = 0;
		while (j<3)
		{
			std::cout << array[i][j] << "  ";
			j++;
		}
		std::cout << std::endl;
		i++;
	}
	temp = determinent(array);
	return temp;
}
void comp_copy(mpreal array[][3], mpreal array1[][3])
{
	//traspasa los valores de la matriz de resultados a la matriz de valores
	int rows = 0, col = 0;
	while (rows<3)
	{
		col = 0;
		while (col<3)
		{
			array[rows][col] = array1[rows][col];
			col++;
		}
		rows++;
	}
}
mpreal determinent4(mpreal array[][4]) {
	int i, j, k;
	aux = 0;
	std::vector<std::vector<mpreal> > matrix(3, std::vector<mpreal>(3));
	mpreal matrixaux[3][3];
	for (i = 0; i<4; i++) {
		matrix = newdet(array, i);
		for (j = 0; j<3; j++) {
			for (k = 0; k<3; k++) {
				matrixaux[j][k] = matrix[j][k];
			}
		}
		aux = aux + pow(-1.0, (mpreal)i)*array[0][i] * (determinent(matrixaux));
	}
	std::cout << "\nDeterminante de la matrix 4x4 es " << aux << "\n\n";
	return aux;
}
std::vector<std::vector<mpreal> > newdet(mpreal array[][4], int col) {
	std::vector<std::vector<mpreal> > matrix(3, std::vector<mpreal>(3));
	int cont = 0, i, j;
	for (i = 1; i<4; i++) {
		for (j = 0; j<4; j++) {
			if (j != col) {
				matrix[i - 1][cont] = array[i][j];
				cont++;
			}
		}
		cont = 0;
	}
	return matrix;
}
void comp_copy4(mpreal array[][4], mpreal array1[][4])
{
	//traspasa los valores de la matriz de resultados a la matriz de valores
	int rows = 0, col = 0;
	while (rows<4)
	{
		col = 0;
		while (col<4)
		{
			array[rows][col] = array1[rows][col];
			col++;
		}
		rows++;
	}
}
mpreal copy4(mpreal array[][4], mpreal array1[][1], int a)
{
	//traspasa los valores de la matriz de resultados a la matriz de valores
	int col = 0;
	temp = 0;
	while (col<4)
	{
		array[col][a] = array1[col][0];
		col++;
	}
	int i = 0, j = 0;
	while (i<4)
	{
		j = 0;
		while (j<4)
		{
			std::cout << array[i][j] << "  ";
			j++;
		}
		std::cout << std::endl;
		i++;
	}
	std::cout << "\n";
	temp = determinent4(array);

	return temp;
}
void input4(mpreal array[][4], mpreal array1[][1])
{
	//traspasa los valores de la matriz X Y Z
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			array[i][j] = (mpreal)matrixChi[i][j];
		}
	}
	//traspasa los resultados de cada ecuacion X + Y + Z = RESULTADO
	for (int i = 0; i < 4; i++) {
		array1[i][0] = (mpreal)matrixChi[i][4];
	}
}